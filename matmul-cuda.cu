#include <iostream>
#include <ranges>

#include <hipblas.h>
#include <eigen3/Eigen/Dense>
#include <thrust/device_vector.h>

#include "utility.h"

constexpr auto M = 1'001;
constexpr auto N = 1'002;
constexpr auto K = 1'003;

auto matmul(hipblasHandle_t &handle, float *x_, float *y_, float *z_, int m, int n, int k) {
  const auto alpha = 1.0f;
  const auto beta  = 0.0f;

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, x_, m, y_, k, &beta, z_, m);  // GEneral Matrix-Matrix multiplication。
}

int main(int argc, char** argv) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  srand(0);

  const auto x = static_cast<Eigen::MatrixXf>(Eigen::MatrixXf::Random(M, K));
  const auto y = static_cast<Eigen::MatrixXf>(Eigen::MatrixXf::Random(K, N));

  auto x_ = thrust::device_vector<float>(M * K);
  auto y_ = thrust::device_vector<float>(K * N);

  hipblasSetMatrix(M, K, sizeof(float), x.transpose().data(), M, x_.data().get(), M);  // Eigenはrow-majorにも対応しているのでこのやり方は無駄なのだけど、とりあえずtranspose()で。
  hipblasSetMatrix(K, N, sizeof(float), y.transpose().data(), K, y_.data().get(), K);

  hipDeviceSynchronize();

  for (const auto &i : std::views::iota(0, 6)) {
    auto z_ = thrust::device_vector<float>(M * N, 0.0f);

    const auto &[duration, _] = duration_and_result([&] {
      matmul(handle, x_.data().get(), y_.data().get(), z_.data().get(), M, N, K);

      hipDeviceSynchronize();

      return 0;
    });

    if (i == 0) {
      continue;
    }

    hipDeviceSynchronize();

    std::cout << duration << "\t" << z_[0] << "\t" << z_[1] << std::endl;
  }

  hipblasDestroy(handle);

  return 0;
}
